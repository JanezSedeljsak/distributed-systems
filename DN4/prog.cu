#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"
#include "stb_image.h"
#include "stb_image_write.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION


#define COLOR_CHANNELS 4
#define BLOCK_SIZE 16

/**
 * module load CUDA/10.1.243-GCC-8.3.0
 * nvcc -o prog.out prog.cu
 * srun --reservation=fri --gpus=1 ./prog.out helmet_in.png helmet_out.png
*/


__device__ inline unsigned char getIntensity(const unsigned char *image, int row, int col,
                                             int channel, int height, int width, int cpp)
{
    if (col < 0 || col >= width)
        return 0;
    if (row < 0 || row >= height)
        return 0;
    return image[(row * width + col) * cpp + channel];
}


// CUDA kernel for image sharpening. Each thread computes one output pixel
__global__ void sharpen(const unsigned char *imageIn, unsigned char *imageOut, const int width, const int height, const int cpp)
{
    // Get pixel
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        for (int c = 0; c < cpp; c++)
        {
            unsigned char px01 = getIntensity(imageIn, y - 1, x, c, height, width, cpp);
            unsigned char px10 = getIntensity(imageIn, y, x - 1, c, height, width, cpp);
            unsigned char px11 = getIntensity(imageIn, y, x, c, height, width, cpp);
            unsigned char px12 = getIntensity(imageIn, y, x + 1, c, height, width, cpp);
            unsigned char px21 = getIntensity(imageIn, y + 1, x, c, height, width, cpp);

            short pxOut = (5 * px11 - px01 - px10 - px12 - px21);
            pxOut = MIN(pxOut, 255);
            pxOut = MAX(pxOut, 0);
            imageOut[(y * width + x) * cpp + c] = (unsigned char)pxOut;
        }
    }
}

int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        printf("USAGE: prog input_image output_image\n");
        exit(EXIT_FAILURE);
    }
    
    char szImage_in_name[255];
    char szImage_out_name[255];

    snprintf(szImage_in_name, 255, "%s", argv[1]);
    snprintf(szImage_out_name, 255, "%s", argv[2]);

    // Load image from file and allocate space for the output image
    int width, height, cpp;
    unsigned char *h_imageIn = stbi_load(szImage_in_name, &width, &height, &cpp, COLOR_CHANNELS);
    cpp = COLOR_CHANNELS;

    if (h_imageIn == NULL)
    {
        printf("Error reading loading image %s!\n", szImage_in_name);
        exit(EXIT_FAILURE);
    }
    printf("Loaded image %s of size %dx%d.\n", szImage_in_name, width, height);
    const size_t datasize = width * height * cpp * sizeof(unsigned char);
    unsigned char *h_imageOut = (unsigned char *)malloc(datasize);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(ceil(width / blockSize.x), ceil(height / blockSize.y));

    unsigned char *d_imageIn;
    unsigned char *d_imageOut;

    // Allocate device memory for images
    checkCudaErrors(hipMalloc(&d_imageIn, datasize));
    checkCudaErrors(hipMalloc(&d_imageOut, datasize));

    // Copy input image to device
    checkCudaErrors(hipMemcpy(d_imageIn, h_imageIn, datasize, hipMemcpyHostToDevice));

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Execute the kernel
    hipEventRecord(start);
    sharpen<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height, cpp);
    getLastCudaError("sharpen() execution failed\n");
    hipEventRecord(stop);

    // Copy image back to host
    checkCudaErrors(hipMemcpy(h_imageOut, d_imageOut, datasize, hipMemcpyDeviceToHost));

    // Wait for the event to finish
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel Execution time is: %0.3f milliseconds \n", milliseconds);

    // Retrieve output file type
    char szImage_out_name_temp[255];
    strncpy(szImage_out_name_temp, szImage_out_name, 255);
    char *token = strtok(szImage_out_name_temp, ".");
    char *FileType = NULL;
    while (token != NULL)
    {
        FileType = token;
        token = strtok(NULL, ".");
    }
    // Write output image to file
    if (!strcmp(FileType, "png"))
        stbi_write_png(szImage_out_name, width, height, cpp, h_imageOut, width * cpp);
    else if (!strcmp(FileType, "jpg"))
        stbi_write_jpg(szImage_out_name, width, height, cpp, h_imageOut, 100);
    else if (!strcmp(FileType, "bmp"))
        stbi_write_bmp(szImage_out_name, width, height, cpp, h_imageOut);
    else
        printf("Error: Unknown image format %s! Only png, bmp, or bmp supported.\n", FileType);

    // Release device memory
    checkCudaErrors(hipFree(d_imageIn));
    checkCudaErrors(hipFree(d_imageOut));

    // Clean up the two events
	hipEventDestroy(start);
	hipEventDestroy(stop);
    
    // Release host memory
    free(h_imageIn);
    free(h_imageOut);

    return 0;
}
