#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#define COLOR_CHANNELS 1
#define BLOCK_SIZE 16
#define GRAYLEVELS 256
#define DESIRED_NCHANNELS 1

// #define LOGGER
#define PERF
#define OPTIMIZED
#define PERF_STEPS // requires PERF flag to be activated

typedef unsigned long long ULL;
typedef unsigned long UL;
typedef unsigned char UC;

/**
 * module load CUDA/10.1.243-GCC-8.3.0
 * nvcc -o histogram.out histogram.cu
 * srun --reservation=fri --gpus=1 ./histogram.out images/500.jpg out/500.jpg
 */

#ifndef OPTIMIZED

__global__ void KERNEL_CalculateHistogram(const UC *image, const int width, const int height, ULL *histogram)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < height && x < width)
    {
        int hist_index = image[y * width + x];
        atomicAdd(histogram + hist_index, 1);
    }
}

__global__ void KERNEL_CalculateCDF(ULL *histogram, ULL *cdf)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x, i;
    ULL sum = 0;

    for (i = 0; i <= x; ++i)
    {
        sum += histogram[i];
    }

    cdf[x] = sum;
}

__global__ void KERNEL_findMin(const ULL *cdf, ULL *min_ptr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (cdf[x] > 0)
    {
        ULL prev = ULONG_LONG_MAX;
        while (cdf[x] < prev)
        {
            prev = atomicCAS(min_ptr, prev, cdf[x]);
        }
    }
}

#else

//__global__ void KERNEL_CalculateHistogram(const UC *image, const int width, const int height, ULL *histogram)
//{
//    // set shared memory to zeros with first thread
//    __shared__ ULL shared[GRAYLEVELS];
//    int i, init_thread = threadIdx.x == 0 && threadIdx.y == 0;
//    if (init_thread) 
//    {
//        for (i = 0; i < GRAYLEVELS; ++i)
//            shared[i] = 0;
//    }
//
//    __syncthreads();
//
//    // add value to shared memory with atomic operation
//    int x = blockIdx.x * blockDim.x + threadIdx.x;
//    int y = blockIdx.y * blockDim.y + threadIdx.y;
//    if (y < height && x < width) 
//    {
//        int hist_index = image[y * width + x];
//        atomicAdd(shared + hist_index, 1);
//    }
//
//    __syncthreads();
//
//    // copy values from shared to histogram
//    if (init_thread) 
//    {
//        for (i = 0; i < GRAYLEVELS; ++i)
//            atomicAdd(histogram + i, shared[i]);
//    }
//}

__global__ void KERNEL_CalculateHistogram(const UC *image, const int width, const int height, ULL *histogram)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < height && x < width)
    {
        int hist_index = image[y * width + x];
        atomicAdd(histogram + hist_index, 1);
    }
}

__global__ void KERNEL_CalculateCDF(ULL *histogram, ULL *cdf)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x, i;
    cdf[x] = histogram[x];

    for (i = 1; i < GRAYLEVELS; i *= 2)
    {
        if (threadIdx.x >= i)
            cdf[threadIdx.x] += cdf[threadIdx.x - i];
    }
}

__global__ void KERNEL_findMin(const ULL *cdf, ULL *min_ptr)
{
    __shared__ ULL shared[GRAYLEVELS];
    int x = blockIdx.x * blockDim.x + threadIdx.x, i;
    shared[threadIdx.x] = cdf[x] > 0 ? cdf[x] : ULONG_LONG_MAX;
    __syncthreads();

    for (i = blockDim.x / 2; i > 0; i /= 2)
    {
        if (threadIdx.x < i && shared[threadIdx.x] > shared[threadIdx.x + i])
            shared[threadIdx.x] = shared[threadIdx.x + i];

        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        ULL prev = ULONG_LONG_MAX;
        while (*shared < prev)
            prev = atomicCAS(min_ptr, prev, *shared);
    }
}

#endif

__device__ inline UC scale(UL cdf, UL cdfmin, UL imageSize)
{
    float scale = (float)(cdf - cdfmin) / (float)(imageSize - cdfmin);
    return (int)round(scale * (float)(GRAYLEVELS - 1));
}

__global__ void KERNEL_Equalize(const UC *image_in, UC *image_out, const int width, const int height, const ULL *cdf, const ULL *cdfmin)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    const UL imageSize = width * height;

    if (y < height && x < width)
    {
        image_out[y * width + x] = scale(cdf[image_in[y * width + x]], *cdfmin, imageSize);
    }
}

int main(int argc, char **argv)
{
    if (argc < 3)
    {
        printf("Error: Missing 2 image params\n");
        exit(EXIT_FAILURE);
    }

    // Read image from file
    int width, height, cpp;

    // read only DESIRED_NCHANNELS channels from the input image:
    UC *imageIn = stbi_load(argv[1], &width, &height, &cpp, DESIRED_NCHANNELS);
    if (imageIn == NULL)
    {
        printf("Error: loading image\n");
        return 1;
    }

    #ifdef LOGGER
        printf("Loaded image W= %d, H = %d, actual cpp = %d \n", width, height, cpp);
    #endif

    const size_t img_size = width * height * sizeof(UC);
    const size_t hist_size = GRAYLEVELS * sizeof(ULL);
    const size_t ull_size = sizeof(ULL);

    // Allocate memory for raw output image data
    UC *imageOut = (UC *)malloc(img_size);

    // Allocate memory for cuda
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(ceil(width / blockSize.x), ceil(height / blockSize.y));

    UC *d_imageIn;
    UC *d_imageOut;
    ULL *d_histogram;
    ULL *d_cdf;
    ULL *d_cdfmin;
    ULL max_value = ULONG_LONG_MAX;

    checkCudaErrors(hipMalloc(&d_imageIn, img_size));
    checkCudaErrors(hipMalloc(&d_imageOut, img_size));
    checkCudaErrors(hipMalloc(&d_histogram, hist_size));
    checkCudaErrors(hipMalloc(&d_cdf, hist_size));
    checkCudaErrors(hipMalloc(&d_cdfmin, ull_size));

    // Copy image CPU -> CUDA and set every cell in histogram and cdf to 0
    checkCudaErrors(hipMemcpy(d_imageIn, imageIn, img_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_histogram, 0, hist_size));
    checkCudaErrors(hipMemset(d_cdf, 0, hist_size));
    checkCudaErrors(hipMemcpy(d_cdfmin, &max_value, ull_size, hipMemcpyHostToDevice));

    // Create CUDA events
    hipEvent_t start, stop, start_1, start_2, start_3, start_4, stop_1, stop_2, stop_3, stop_4;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&start_1);
    hipEventCreate(&stop_1);
    hipEventCreate(&start_2);
    hipEventCreate(&stop_2);
    hipEventCreate(&start_3);
    hipEventCreate(&stop_3);
    hipEventCreate(&start_4);
    hipEventCreate(&stop_4);

    // Histogram equalization steps:
    hipEventRecord(start);
    hipEventRecord(start_1);

    // 1. Create the histogram for the input grayscale image.
    KERNEL_CalculateHistogram<<<gridSize, blockSize>>>(d_imageIn, width, height, d_histogram);

    hipEventRecord(stop_1);
    hipEventRecord(start_2);

    // 2. Calculate the cumulative distribution histogram.
    dim3 blocks256(GRAYLEVELS);
    dim3 gridSizeHist(1);
    KERNEL_CalculateCDF<<<gridSizeHist, blocks256>>>(d_histogram, d_cdf);

    hipEventRecord(stop_2);
    hipEventRecord(start_3);


    // 3. Calculate the OPTIMIZED gray-level values through the general histogram equalization formula and assign OPTIMIZED pixel values
    dim3 gridSizeMin((GRAYLEVELS + blocks256.x - 1) / blocks256.x);
    KERNEL_findMin<<<gridSizeMin, blocks256>>>(d_cdf, d_cdfmin);

    hipEventRecord(stop_3);
    hipEventRecord(start_4);

    #ifdef LOGGER
        checkCudaErrors(hipMemcpy(&max_value, d_cdfmin, ull_size, hipMemcpyDeviceToHost));
        printf("First greater than 0: %llu\n", max_value);
    #endif

    KERNEL_Equalize<<<gridSize, blockSize>>>(d_imageIn, d_imageOut, width, height, d_cdf, d_cdfmin);

    hipEventRecord(stop_4);
    hipEventRecord(stop);

    // Copy data CUDA -> CPU
    checkCudaErrors(hipMemcpy(imageOut, d_imageOut, img_size, hipMemcpyDeviceToHost));

    // Wait for the event to finish
    hipEventSynchronize(stop);

    #ifdef PERF
        float milliseconds = 0;

        #ifdef PERF_STEPS
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Global: %0.3f\n", milliseconds);

            hipEventElapsedTime(&milliseconds, start_1, stop_1);
            printf("1: %0.3f\n", milliseconds);

            hipEventElapsedTime(&milliseconds, start_2, stop_2);
            printf("2: %0.3f\n", milliseconds);

            hipEventElapsedTime(&milliseconds, start_3, stop_3);
            printf("3: %0.3f\n", milliseconds);

            hipEventElapsedTime(&milliseconds, start_4, stop_4);
            printf("4: %0.3f\n", milliseconds);
        #else
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("%0.3f\n", milliseconds);
        #endif
    #endif

    // Create output image:
    stbi_write_jpg(argv[2], width, height, DESIRED_NCHANNELS, imageOut, 100);
    // stbi_write_png("out.png", width, height, DESIRED_NCHANNELS, imageOut, width * DESIRED_NCHANNELS);

    // Free CUDA memory
    checkCudaErrors(hipFree(d_imageIn));
    checkCudaErrors(hipFree(d_imageOut));
    checkCudaErrors(hipFree(d_histogram));
    checkCudaErrors(hipFree(d_cdf));
    checkCudaErrors(hipFree(d_cdfmin));

    // Clean up the two events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(start_1);
    hipEventDestroy(stop_1);
    hipEventDestroy(start_2);
    hipEventDestroy(stop_2);
    hipEventDestroy(start_3);
    hipEventDestroy(stop_3);
    hipEventDestroy(start_4);
    hipEventDestroy(stop_4);

    // Free memory
    free(imageIn);
    free(imageOut);

    return 0;
}
